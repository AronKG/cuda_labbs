
#include <hip/hip_runtime.h>
#include <iostream>  // Include the standard input-output library
#include <math.h>    // Include math functions for checking errors

// CUDA kernel function to multiply elements of two arrays
__global__ void multKernel(int n, float* a, float* b, float* c) 
{ 
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the stride for accessing elements based on total threads in grid
    int stride = blockDim.x * gridDim.x;

    // Loop over array elements in strides so each thread processes multiple elements
    for (int i = index; i < n; i += stride) {
        c[i] = a[i] * b[i];  // Multiply elements of arrays 'a' and 'b' and store in 'c'
    }
}

int main() {
    int N = 1 << 24;  // Define the size of the arrays (2^24 elements)
    
    // Declare pointers for host arrays
    float *h_a, *h_b, *h_c;
    
    // Declare pointers for device arrays
    float *d_a, *d_b, *d_c;

    // Allocate memory for host arrays
    h_a = new float[N];
    h_b = new float[N];
    h_c = new float[N];

    // Allocate memory for device arrays on the GPU
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Initialize the host arrays 'h_a' and 'h_b' with values
    for (int i = 0; i < N; i++) {
        h_a[i] = 2.0f;  // Set each element of 'h_a' to 2.0
        h_b[i] = 3.0f;  // Set each element of 'h_b' to 3.0
    }

    // Copy the initialized data from host arrays to device arrays
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Define the block size and calculate the number of blocks needed
    int blockSize = 256;  // Number of threads per block
    int numBlocks = (N + blockSize - 1) / blockSize;  // Total number of blocks

    // Launch the kernel with the specified number of blocks and threads per block
    multKernel<<<numBlocks, blockSize>>>(N, d_a, d_b, d_c);

    // Copy the result from device memory back to host memory
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check the result for any errors by comparing with the expected value (6.0f)
    float maxError = 0.0f;  // Variable to store the maximum error found
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(h_c[i] - 6.0f));  // Calculate error and update maxError if needed
    }

    // Print the maximum error found in the results
    std::cout << "Max error: " << maxError << std::endl;

    // Free allocated device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free allocated host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;  // End of the program
}
