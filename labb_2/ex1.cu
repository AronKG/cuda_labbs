
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to multiply two arrays
__global__ void multKernel(int n, float* a, float* b, float* c) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] * b[i]; // Perform element-wise multiplication
    }
}

int main() {
    int N = 1 << 24; // Define the size of the arrays (2^24 elements)
    float *h_a, *h_b, *h_c; // Host pointers
    float *d_a, *d_b, *d_c; // Device pointers

    // Allocate host memory
    h_a = new float[N]; // Host array A
    h_b = new float[N]; // Host array B
    h_c = new float[N]; // Host array C for results

    // Allocate device memory
    hipMalloc(&d_a, N * sizeof(float)); // Device array A
    hipMalloc(&d_b, N * sizeof(float)); // Device array B
    hipMalloc(&d_c, N * sizeof(float)); // Device array C

    // Initialize host data
    for (int i = 0; i < N; i++) {
        h_a[i] = 2.0f; // Fill A with 2.0
        h_b[i] = 3.0f; // Fill B with 3.0
    }

    // Copy data from host to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    multKernel<<<1, 1>>>(N, d_a, d_b, d_c); // Execute kernel with 1 block of 1 thread

    // Copy result back to host
    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Check result for errors (all values should be 6.0f)
    float maxError = 0.0f; // Initialize max error
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(h_c[i] - 6.0f)); // Check max error from expected result

    std::cout << "Max error: " << maxError << std::endl; // Print max error

    // Clean up
    hipFree(d_a); // Free device memory A
    hipFree(d_b); // Free device memory B
    hipFree(d_c); // Free device memory C
    delete[] h_a;  // Free host memory A
    delete[] h_b;  // Free host memory B
    delete[] h_c;  // Free host memory C

    return 0; // Return success
}
