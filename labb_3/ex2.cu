#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>  // Include the header for video source handling
#include <jetson-utils/videoOutput.h>  // Include the header for video output handling

// CUDA kernel function to convert an RGB image to grayscale
__global__ void rgb2grayKernel(uchar4* image, int width, int height)
{ 
    // Calculate the total number of pixels in the image
    long int size = width * height; 
    // Calculate the index of the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the stride for accessing pixels in a loop
    int stride = blockDim.x * gridDim.x;
    
    // Loop over the image pixels in strides to ensure all pixels are processed
    for (int i = index; i < size; i += stride) {
        // Calculate the grayscale value using the luminosity method
        unsigned char gray = (0.299 * image[i].x) + (0.587 * image[i].y) + (0.114 * image[i].z); 
        
        // Set the RGB values of the pixel to the grayscale value
        image[i].x = gray; // Red channel
        image[i].y = gray; // Green channel
        image[i].z = gray; // Blue channel
        // Alpha channel (image[i].w) is unchanged
    }
}

int main(int argc, char** argv)
{
    // Create input/output streams for the video source and output
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    
    // Check if the input video source was successfully created
    if (!input)
        return 0; // Exit if input cannot be created

    // Capture/display loop to continuously process frames
    while (true)
    {
        uchar4* image = NULL; // Pointer to the image buffer
        int status = 0; // Status variable for video source (OK, TIMEOUT, EOS, ERROR)
        
        // Capture an image from the video source with a 1000ms timeout
        if (!input->Capture(&image, 1000, &status)) 
        {
            // Check if the capture timed out and continue the loop if so
            if (status == videoSource::TIMEOUT)
                continue;
            break; // End of stream (EOS), break the loop
        }

        // Launch the kernel to convert the image to grayscale
        rgb2grayKernel<<<16, 1024>>>(image, input->GetWidth(), input->GetHeight());

        // If the output stream is valid, render the processed image
        if (output != NULL)
        {
            output->Render(image, input->GetWidth(), input->GetHeight());
            // Update the status bar with the current frame dimensions and FPS
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            // Check if the user has quit the streaming
            if (!output->IsStreaming())
                break; // Exit the loop if not streaming
        }
    }
}
