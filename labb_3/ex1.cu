#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>  // Include the header for video input source functionality
#include <jetson-utils/videoOutput.h>  // Include the header for video output display functionality


int main( int argc, char** argv )
{
    // Create input stream for video capture, using command-line arguments for configuration
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    
    // Create output stream for video display, also using command-line arguments for configuration
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    
    // If input stream couldn't be created, exit the program
    if ( !input )
        return 0;

    // Main loop for capturing and displaying frames
    while (true)
    {
        uchar4* image = NULL;  // Initialize a pointer to hold the captured image; uchar4 means each pixel has 4 channels (RGBA)
        int status = 0;  // Variable to track capture status (OK, TIMEOUT, EOS, ERROR)
        
        // Attempt to capture a frame with a 1000ms timeout; store the status
        if ( !input->Capture(&image, 1000, &status) )
        {
            // If capture timed out, continue the loop to try capturing again
            if (status == videoSource::TIMEOUT)
                continue;
            
            // If any other status (e.g., EOS), exit the loop to end the program
            break;
        }
        
        // Check if the output stream is initialized
        if ( output != NULL )
        {
            // Render the captured image on the output display with its dimensions
            output->Render(image, input->GetWidth(), input->GetHeight());
            
            // Update the status bar with the resolution and current frames per second (FPS)
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            
            // If the user has closed the output stream, exit the loop
            if (!output->IsStreaming())
                break;
        }
    }
}
