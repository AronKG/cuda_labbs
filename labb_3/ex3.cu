#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>  // Include the header for video source handling
#include <jetson-utils/videoOutput.h>  // Include the header for video output handling

// CUDA kernel function to convert an RGB image to grayscale
__global__ void rgb2grayKernel(uchar4* image, uchar4* output, int width, int height)
{ 
    // Calculate the total number of pixels in the image
    long int size = width * height; 
    // Calculate the index of the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the stride for accessing pixels in a loop
    int stride = blockDim.x * gridDim.x;

    // Loop over the image pixels in strides to ensure all pixels are processed
    for (int i = index; i < size; i += stride) {
        // Calculate the grayscale value using the luminosity method
        unsigned char gray = (0.299 * image[i].x) + (0.587 * image[i].y) + (0.114 * image[i].z); 
        
        // Set the RGB values of the output pixel to the grayscale value
        output[i].x = gray; // Red channel
        output[i].y = gray; // Green channel
        output[i].z = gray; // Blue channel
        // The alpha channel (output[i].w) remains unchanged
    }
}

int main(int argc, char** argv)
{
    // Create input and output streams for video handling
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(2)); // Create a second output for grayscale

    // Check if the input video source was successfully created
    if (!input)
        return 0; // Exit if input cannot be created

    // Allocate device memory for the output image
    uchar4* d_output;
    size_t imageSize = input->GetWidth() * input->GetHeight() * sizeof(uchar4);
    hipMalloc((void**)&d_output, imageSize); // Allocate memory on the GPU for output image

    // Capture/display loop to continuously process frames
    while (true)
    {
        uchar4* image = NULL; // Pointer to the image buffer
        int status = 0; // Status variable for video source (OK, TIMEOUT, EOS, ERROR)
        
        // Capture an image from the video source with a 1000ms timeout
        if (!input->Capture(&image, 1000, &status)) 
        {
            // Check if the capture timed out and continue the loop if so
            if (status == videoSource::TIMEOUT)
                continue; // If timeout, skip to the next iteration
            break; // End of stream (EOS), break the loop
        }

        // Launch the kernel to convert the image to grayscale
        rgb2grayKernel<<<16, 1024>>>(image, d_output, input->GetWidth(), input->GetHeight());

        // If the primary output stream is valid, render the original image
        if (output != NULL)
        {
            output->Render(image, input->GetWidth(), input->GetHeight());

            // Update the status bar with the current frame dimensions and FPS
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            // Check if the user has quit the streaming
            if (!output->IsStreaming())
                break; // Exit the loop if not streaming
        }

        // If the second output stream is valid, render the grayscale image
        if (output2 != NULL)
        {
            output2->Render(d_output, input->GetWidth(), input->GetHeight());

            // Update the status bar for the second output stream
            char str[256];
            sprintf(str, "Grayscale Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output2->GetFrameRate());
            output2->SetStatus(str);

            // Check if the user has quit the streaming
            if (!output2->IsStreaming())
                break; // Exit the loop if not streaming
        }
    }

    // Free allocated device memory
    hipFree(d_output); // Release the allocated GPU memory for the output image
}
