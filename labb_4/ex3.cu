#include "hip/hip_runtime.h"

#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <stdio.h> 
__global__ void rgb2grayKernel(uchar4* image, uchar4* output, int width, int height)
{ 
    
    long int size = width*height; 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
    
        unsigned char gray = (0.299*image[i].x) + (0.587*image[i].y) + (0.114*image[i].z); 
        output[i].x  = gray; 
        output[i].y = gray; 
        output[i].z = gray; 
    }

}

__global__ void calcHistogramKernel(uchar4* d_output, int* histogram, int width, int height)
{ 
    __shared__ int histo_local[256];


    long int size = width*height; 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
 
    if(index < 256)
    {
        histogram[index]= 0;
        histo_local[index] = 0; 
    }

    __syncthreads(); 
    
    for (int i = index; i < size; i += stride) {
    
        unsigned char gray = d_output[i].x; 
        atomicAdd(&histo_local[gray], 1);
    }

    __syncthreads(); 

    if (threadIdx.x < 256) {
        atomicAdd(&histogram[threadIdx.x], histo_local[threadIdx.x]);
    }

}

__global__ void plotHistogramKernel(uchar4* image, int* histogram, int width, int height, int max_freq)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        uchar4 white_pixel = make_uchar4(255, 255, 255, 255);
        uchar4 black_pixel = make_uchar4(0, 0, 0, 255);

        if (index < 256)
        {
            int freq = histogram[index] * 256 / max_freq;
            for (int i = 0; i < 256; i++)
            {
                int row = height - i - 1;
              if (i <= freq)
               {
                  image[row * width + 2*index] = white_pixel;
                  image[row * width + 2*index+1] = white_pixel;
               }
              else
                 {
                   image[row * width + 2*index] = black_pixel;
                   image[row * width + 2*index+1] = black_pixel;
                 }
            }
        }
    }


int main( int argc, char** argv )
{
    int max_freq = 20000; 
    // create input/output streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(1));

    if ( !input )
    return 0;

    uchar4* d_output;
        size_t imageSize = input->GetWidth() * input->GetHeight() * sizeof(uchar4);

    hipMalloc((void**)&d_output, imageSize);
    int* histogram; 
    hipMalloc(&histogram, 256*sizeof(int));
    int host_histo[256];

// capture/display loop
    while (true)
    {
        uchar4* image = NULL; // can be uchar3, uchar4, float3, float4
        int status = 0; // see videoSource::Status (OK, TIMEOUT, EOS,ERROR)
        if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
        {
            if (status == videoSource::TIMEOUT)
            continue;
            break; // EOS
        }
        rgb2grayKernel<<<16, 1024>>>(image, d_output, input->GetWidth(), input->GetHeight());
        calcHistogramKernel<<<16, 1024>>>(d_output, histogram, input->GetWidth(), input->GetHeight());

        plotHistogramKernel<<<1, 256>>>(d_output,histogram,input->GetWidth(), input->GetHeight(), max_freq);

        // Copy result from device to host
        hipMemcpy(host_histo, histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);

        int sum = 0; 
        for (int i = 0; i < 256 ; i++)
        {
            sum += host_histo[i]; 

        }
        printf("%d\n",sum); 

        if ( output != NULL )
        {
            output->Render(image, input->GetWidth(), input->GetHeight());


            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            if (!output->IsStreaming()) // check if the user quit
            break;
      }

      if ( output2 != NULL )
      {

          output2->Render(d_output, input->GetWidth(), input->GetHeight());

          // Update status bar
          char str[256];
          sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
          input->GetHeight(), output->GetFrameRate());
          output->SetStatus(str);
          if (!output->IsStreaming()) // check if the user quit
          break;
    }
    }

}


