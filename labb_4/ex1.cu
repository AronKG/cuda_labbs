#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>
#include <stdio.h>

// Kernel to convert RGB image to grayscale
__global__ void rgb2grayKernel(uchar4* image, uchar4* output, int width, int height)
{ 
    // Calculate total image size
    long int size = width * height; 

    // Determine pixel index and stride for parallel processing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over the pixels handled by this thread
    for (int i = index; i < size; i += stride) {
        // Compute grayscale intensity using weighted average of RGB values
        unsigned char gray = (0.299 * image[i].x) + (0.587 * image[i].y) + (0.114 * image[i].z); 
        output[i].x = gray; 
        output[i].y = gray; 
        output[i].z = gray; 
    }
}

// Kernel to calculate histogram for grayscale image
__global__ void calcHistogramKernel(uchar4* d_output, int* histogram, int width, int height)
{ 
    // Calculate total image size
    long int size = width * height; 

    // Determine pixel index and stride for parallel processing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize histogram bins to zero (one bin per thread, up to 256 threads)
    if (index < 256) {
        histogram[index] = 0; 
    }

    // Synchronize threads to ensure histogram initialization is complete
    __syncthreads(); 

    // Update histogram by counting grayscale intensities
    for (int i = index; i < size; i += stride) {
        unsigned char gray = d_output[i].x; 
        atomicAdd(&histogram[gray], 1); // Atomic addition to avoid race conditions
    }
}

int main(int argc, char** argv)
{
    // Create input and output video streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(1));

    if (!input)
        return 0;

    uchar4* d_output; // Output for grayscale image on device
    size_t imageSize = input->GetWidth() * input->GetHeight() * sizeof(uchar4);

    // Allocate memory for grayscale image and histogram on device
    hipMalloc((void**)&d_output, imageSize);
    int* histogram; 
    hipMalloc(&histogram, 256 * sizeof(int));
    int host_histo[256]; // Array to store histogram on host

    // Capture and display loop
    while (true)
    {
        uchar4* image = NULL; // Pointer to captured image
        int status = 0; // Status variable for videoSource capture
        if (!input->Capture(&image, 1000, &status)) // Capture image with 1000ms timeout
        {
            if (status == videoSource::TIMEOUT)
                continue;
            break; // End of stream or error
        }

        // Launch grayscale conversion and histogram calculation kernels
        rgb2grayKernel<<<16, 1024>>>(image, d_output, input->GetWidth(), input->GetHeight());
        calcHistogramKernel<<<16, 1024>>>(d_output, histogram, input->GetWidth(), input->GetHeight());

        // Copy histogram from device to host
        hipMemcpy(host_histo, histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);

        // Sum all histogram values to verify total pixel count (sanity check)
        int sum = 0; 
        for (int i = 0; i < 256; i++) {
            sum += host_histo[i]; 
        }
        printf("%d\n", sum); // Print the sum for verification

        // Display original image
        if (output != NULL) {
            output->Render(image, input->GetWidth(), input->GetHeight());

            // Update status bar with resolution and frame rate
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            if (!output->IsStreaming()) // Check if the user quit
                break;
        }

        // Display grayscale image
        if (output2 != NULL) {
            output2->Render(d_output, input->GetWidth(), input->GetHeight());

            // Update status bar with resolution and frame rate
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            if (!output->IsStreaming()) // Check if the user quit
                break;
        }
    }
}
