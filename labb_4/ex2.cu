#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>  // Library for video input
#include <jetson-utils/videoOutput.h>  // Library for video output
#include <stdio.h> 

// Kernel to convert an RGB image to grayscale
__global__ void rgb2grayKernel(uchar4* image, uchar4* output, int width, int height)
{ 
    long int size = width * height;  // Total number of pixels
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique index for each thread
    int stride = blockDim.x * gridDim.x;  // Define stride for parallel processing

    // Each thread processes pixels in strides to cover the whole image
    for (int i = index; i < size; i += stride) {
        // Grayscale formula using luminance coefficients for RGB channels
        unsigned char gray = (0.299 * image[i].x) + (0.587 * image[i].y) + (0.114 * image[i].z);
        output[i].x = gray;  // Set red channel to grayscale value
        output[i].y = gray;  // Set green channel to grayscale value
        output[i].z = gray;  // Set blue channel to grayscale value
    }
}

// Kernel to calculate the histogram of the grayscale image
__global__ void calcHistogramKernel(uchar4* d_output, int* histogram, int width, int height)
{ 
    long int size = width * height;  // Total number of pixels
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Unique index for each thread
    int stride = blockDim.x * gridDim.x;  // Define stride for parallel processing

    // Initialize histogram bins to 0 (only the first 256 threads do this)
    if(index < 256) {
        histogram[index] = 0; 
    }
    __syncthreads();  // Ensure all threads have finished initialization

    // Each thread processes pixels in strides to cover the whole image
    for (int i = index; i < size; i += stride) {
        unsigned char gray = d_output[i].x;  // Access the grayscale value
        atomicAdd(&histogram[gray], 1);  // Use atomic addition to avoid race conditions
    }
}

// Kernel to plot the histogram as a bar graph on the image
__global__ void plotHistogramKernel(uchar4* image, int* histogram, int width, int height, int max_freq)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Unique index for each thread
    uchar4 white_pixel = make_uchar4(255, 255, 255, 255);  // Define white color for bars
    uchar4 black_pixel = make_uchar4(0, 0, 0, 255);  // Define black color for background

    if (index < 256) {  // Only the first 256 threads plot the histogram
        int freq = histogram[index] * 256 / max_freq;  // Scale frequency to fit graph height

        // Draw each bar of the histogram
        for (int i = 0; i < 256; i++) {
            int row = height - i - 1;  // Start from the bottom of the image
            if (i <= freq) {
                // Set pixels to white for the histogram bar
                image[row * width + 2 * index] = white_pixel;
                image[row * width + 2 * index + 1] = white_pixel;
            } else {
                // Set pixels to black for the background
                image[row * width + 2 * index] = black_pixel;
                image[row * width + 2 * index + 1] = black_pixel;
            }
        }
    }
}

int main(int argc, char** argv)
{
    int max_freq = 20000;  // Maximum frequency for histogram scaling

    // Create input and output streams for video
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));  // Video input source
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));  // Video output for original frame
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(1));  // Video output for grayscale and histogram

    if (!input) return 0;  // Exit if input source is not created

    // Allocate memory for grayscale output on the GPU
    uchar4* d_output;
    size_t imageSize = input->GetWidth() * input->GetHeight() * sizeof(uchar4);
    hipMalloc((void**)&d_output, imageSize);

    // Allocate memory for histogram on the GPU
    int* histogram; 
    hipMalloc(&histogram, 256 * sizeof(int));
    int host_histo[256];  // Host array to store histogram results

    // Capture and display loop
    while (true) {
        uchar4* image = NULL;  // Pointer for the captured frame
        int status = 0;  // Status for video capture
        if (!input->Capture(&image, 1000, &status)) {  // Capture frame with 1000 ms timeout
            if (status == videoSource::TIMEOUT) continue;  // Skip if timeout
            break;  // Exit if end of stream (EOS)
        }

        // Launch kernel to convert image to grayscale
        rgb2grayKernel<<<16, 1024>>>(image, d_output, input->GetWidth(), input->GetHeight());

        // Launch kernel to calculate the histogram
        calcHistogramKernel<<<16, 1024>>>(d_output, histogram, input->GetWidth(), input->GetHeight());

        // Launch kernel to plot histogram on grayscale image
        plotHistogramKernel<<<1, 256>>>(d_output, histogram, input->GetWidth(), input->GetHeight(), max_freq);

        // Copy histogram data from device to host
        hipMemcpy(host_histo, histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);

        // Sum the histogram values and print to console
        int sum = 0; 
        for (int i = 0; i < 256 ; i++) {
            sum += host_histo[i]; 
        }
        printf("%d\n", sum); 

        // Render original frame if output stream exists
        if (output != NULL) {
            output->Render(image, input->GetWidth(), input->GetHeight());

            // Update status bar with frame rate info
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            if (!output->IsStreaming()) break;  // Exit if user quits
        }

        // Render grayscale frame with histogram if output2 exists
        if (output2 != NULL) {
            output2->Render(d_output, input->GetWidth(), input->GetHeight());

            // Update status bar with frame rate info
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
                    input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);
            if (!output->IsStreaming()) break;  // Exit if user quits
        }
    }
}
